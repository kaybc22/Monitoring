#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("Error: Failed to get device count: %s\n", hipGetErrorString(err));
        return 1;
    }
#nvcc -o gpu_test gpu_test.cu -gencode arch=compute_100,code=sm_100
    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
        return 1;
    }

    printf("Found %d CUDA-capable device(s):\n", deviceCount);
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    }

    return 0;
}
