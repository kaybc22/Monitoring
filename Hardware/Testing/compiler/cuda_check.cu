#include <iostream>
#include <hip/hip_runtime.h>
int main() {
   int deviceCount = 0;
   hipError_t err = hipGetDeviceCount(&deviceCount);
   if (err != hipSuccess) {
       std::cerr << "hipGetDeviceCount failed: "
<< hipGetErrorString(err) << std::endl;
       return 1;
   }
   std::cout << "Detected " << deviceCount << " CUDA device(s)\n";
   for (int i = 0; i < deviceCount; i++) {
       std::cout << "\nChecking GPU " << i << "...\n";
       // Try to set the device
       err = hipSetDevice(i);
       if (err != hipSuccess) {
           std::cerr << "  Failed to set device " << i << ": "
<< hipGetErrorString(err) << std::endl;
           continue;
       }
       // Query device properties
       hipDeviceProp_t prop;
       err = hipGetDeviceProperties(&prop, i);
       if (err != hipSuccess) {
           std::cerr << "  Failed to get properties: "
<< hipGetErrorString(err) << std::endl;
           continue;
       }
       std::cout << "  Name: " << prop.name
<< " | Global Mem: " << (prop.totalGlobalMem >> 20) << " MB"
<< " | SMs: " << prop.multiProcessorCount << std::endl;
       // Test a small allocation
       void* d_ptr = nullptr;
       err = hipMalloc(&d_ptr, 1024);
       if (err != hipSuccess) {
           std::cerr << "  hipMalloc failed: "
<< hipGetErrorString(err) << std::endl;
       } else {
           std::cout << "  hipMalloc test OK\n";
           hipFree(d_ptr);
       }
   }
   return 0;
}
