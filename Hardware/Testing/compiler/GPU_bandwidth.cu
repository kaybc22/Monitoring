#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int gpuA, gpuB;
    size_t dataMB;

    std::cout << "Enter source GPU ID (1�8): ";
    std::cin >> gpuA;
    std::cout << "Enter destination GPU ID (1�8): ";
    std::cin >> gpuB;
    std::cout << "Enter data size in MB (e.g., 10, 100000): ";
    std::cin >> dataMB;

    gpuA -= 1;
    gpuB -= 1;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (gpuA < 0 || gpuB < 0 || gpuA >= deviceCount || gpuB >= deviceCount) {
        std::cerr << "Invalid GPU IDs. Available GPUs: 1 to " << deviceCount << std::endl;
        return 1;
    }

    int canAccessPeer = 0;
    hipDeviceCanAccessPeer(&canAccessPeer, gpuB, gpuA);
    if (!canAccessPeer) {
        std::cerr << "GPU " << gpuB + 1 << " cannot access GPU " << gpuA + 1 << " via P2P." << std::endl;
        return 1;
    }

    hipSetDevice(gpuB);
    hipDeviceEnablePeerAccess(gpuA, 0);

    size_t dataSize = dataMB * 1024 * 1024; // Convert MB to bytes

    hipSetDevice(gpuA);
    float* src;
    hipMalloc(&src, dataSize);
    hipMemset(src, 1, dataSize);

    hipSetDevice(gpuB);
    float* dst;
    hipMalloc(&dst, dataSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpyPeer(dst, gpuB, src, gpuA, dataSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    double gb = static_cast<double>(dataSize) / (1024.0 * 1024.0 * 1024.0);
    double bandwidth = gb / (ms / 1000.0);

    std::cout << "\n? Data transferred from GPU " << gpuA + 1 << " to GPU " << gpuB + 1 << std::endl;
    std::cout << "?? Transfer size: " << gb << " GB" << std::endl;
    std::cout << "?? Transfer time: " << ms << " ms" << std::endl;
    std::cout << "?? Bandwidth: " << bandwidth << " GB/s\n" << std::endl;

    hipFree(src);
    hipFree(dst);
    hipDeviceDisablePeerAccess(gpuA);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}